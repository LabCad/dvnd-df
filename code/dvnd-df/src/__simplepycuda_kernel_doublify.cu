
		
#include <hip/hip_runtime.h>
#include<stdio.h>
		

struct simplepycuda_grid { int x,y; };

struct simplepycuda_block { int x,y,z; };

__global__ void kernel_doublify( float* a )
		{
			int idx = threadIdx.x + threadIdx.y*4;
			a[idx] *= 2;
		}
	
extern "C" void kernel_loader( float* a , simplepycuda_grid g, simplepycuda_block b, size_t shared, size_t stream) {
//	printf("lets go! grid(%d,%d) block(%d,%d,%d) shared=%lu stream=%lu\n",g.x,g.y,b.x,b.y,b.z,shared,stream);
	dim3 mygrid;  mygrid.x = g.x;  mygrid.y = g.y;
	dim3 myblock; myblock.x = b.x; myblock.y = b.y; myblock.z = b.z;
	kernel_doublify<<<mygrid, myblock, shared, hipStream_t(stream)>>>( a);
hipDeviceSynchronize();
//	printf("finished kernel!");
}


//nvcc --shared __simplepycuda_kernel_doublify.cu  -o __simplepycuda_kernel_doublify.so --compiler-options -fPIC 2> __simplepycuda_kernel_doublify.log
